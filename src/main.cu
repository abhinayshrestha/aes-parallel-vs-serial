#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include "aes_serial.h"
#include "aes_parallel.h"
#include <hip/hip_runtime.h>
#include "timer.h"

int main()
{
    unsigned char message[] = "The AES128 algorithm is a highly efficient and secure cryptographic method designed to encrypt data in both parallel and serial modes, ensuring robust protection against unauthorized access. Utilizing a 128-bit key, this algorithm transforms plaintext into a 128-bit ciphertext through a sequence of intricate operations. To begin, the input message is divided into blocks, each 128 bits in size, which are then processed independently. The key expansion mechanism generates a unique round key for every stage, producing 11 distinct keys for the 10 main encryption rounds and the initial pre-round transformation. These keys are crucial to the algorithm's strength, ensuring that each block is encrypted differently.Each encryption round consists of several transformative steps: SubBytes replaces bytes based on a substitution box, offering non-linearity; ShiftRows rearranges rows to disperse the data further; MixColumns introduces data diffusion across columns, making it difficult to trace original data patterns; and AddRoundKey combines the state with the round key using an XOR operation. In the final round, the MixColumns step is omitted to maintain ciphertext compactness. Parallel processing allows AES128 to encrypt large volumes of data rapidly, while serial execution ensures precise and consistent operations for smaller datasets. This flexibility makes the algorithm suitable for a wide range of applications, including securing online banking, safeguarding personal communications, and protecting classified government information. Its resistance to modern cryptanalytic attacks like differential and linear cryptanalysis underscores its reliability.AES128's adoption across industries highlights its trustworthiness. By breaking messages into blocks and applying layered transformations, it guarantees confidentiality. Whether encrypting a short text or vast datasets, it ensures that sensitive information remains secure, reinforcing its role as a cornerstone of modern data encryption technologies.";
    unsigned char key[] = "thesouthernmisss";
    printf("\n *******************Starting**********************\n");
    printf("Message: %s\n", message);
    printf("Key: %s\n", key);
    printf("\n\n");
    AES_Encryption_Serial(message, key);
    printf("\n\n");
    AES_Encryption_Parallel(message, key);
    return 0;
}